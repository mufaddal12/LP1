#include "hip/hip_runtime.h"
%%cu

#include <iostream>
#include <time.h>
#include <stdexcept>
#include <vector> 
#include <cstdlib>
#include <chrono>

#define r1 512
#define c1 512
#define r2 512
#define c2 512
using namespace std;


__global__ void matmul(int* a,int* b, int* c){
		
	int x = blockIdx.x;
	int y = blockIdx.y;

	c[c2 * y + x] = 0;
	for(int k = 0; k < c1; k++)
		c[c2 * y + x] += (a[c1 * y + k] * b[c2 * k + x]);

}
int main(void){

	srand(time(0));
	int a[r1][c1];
	int b[r2][c2];
	int c[r1][c2];
	

	for(int i = 0; i < r1; i++)
		for(int j = 0; j < c1; j++)
			a[i][j] = rand();

	for(int i = 0; i < r2; i++)
		for(int j = 0; j < c2; j++)
			b[i][j] = rand();

	int *p ,*q, *r;

	hipMalloc((void**)&p,r1 * c1 * sizeof(int));
	hipMalloc((void**)&q,r2 * c2 * sizeof(int));
	hipMalloc((void**)&r,r1 * c2 * sizeof(int));

	hipMemcpy(p,a,r1 * c1 * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(q,b,r1 * c1 * sizeof(int),hipMemcpyHostToDevice);

	dim3 grid(c2,r1); // cols * rows

  auto start = chrono::steady_clock::now();
  matmul<<<grid,1>>>(p,q,r);
  auto end = chrono::steady_clock::now();

	hipMemcpy(c,r, r1 * c2 * sizeof(int),hipMemcpyDeviceToHost);

  cout << "GPU STATISTICS:\n";
  cout << "Time taken: " << chrono::duration_cast<chrono::microseconds>(end - start).count() << " microseconds\n";

	hipFree(p);
	hipFree(q);
	hipFree(r);
	return 0;
}
